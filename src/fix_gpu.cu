#include "hip/hip_runtime.h"
#include "fix_gpu.cuh"
#include "image.hh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <cuda/atomic>

// We tried to implement Brent-Kung scan because the last value of each block (the sum of the block to propagate to the next one)
// is available before the end of the scan, so it would allow the next block to compute its value earlier. However, we were not able
// to make it work.
/*__global__
void compact_scan(int* image, int size, int *blockNb, cuda::std::atomic<char>* flags, int* predicate){

    __shared__ int blockId;
    if (threadIdx.x == 0)
        blockId = atomicAdd(blockNb, 1);
    __syncthreads();

    extern __shared__ int sdata[];

    constexpr int garbage_val = -27;

    int tid = threadIdx.x;
    int i = blockId * blockDim.x + threadIdx.x;

    if (i < size){
        if (image[i] != garbage_val)
            sdata[tid] = 1;
        else
            sdata[tid] = 0;
    }
    __syncthreads();


    int stride = 1;

    while (stride < 2 * blockDim.x){
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index < 2 * blockDim.x && (index - stride) >= 0)
            sdata[index] += sdata[index - stride];
        stride *= 2;
        __syncthreads();
    }
    __syncthreads();

    if (i < size && blockId < gridDim.x - 1)
        predicate[tid + (blockId + 1) * blockDim.x] += sdata[blockDim.x - 1];
    
    __syncthreads();

    if (i < size && blockId == 0 && tid == 0)
        flags[blockId].store('P');
    else if (i < size && tid == 0 && blockId > 0 && flags[blockId - 1].load() == 'P')
        flags[blockId].store('P');

    __syncthreads();

    stride = blockDim.x / 2;
    while (stride > 0){
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < 2 * blockDim.x)
            sdata[index + stride] += sdata[index];
        stride /= 2;
    }

    __syncthreads();

    if (i < size)
        predicate[tid + blockId * blockDim.x] += sdata[tid];

    __syncthreads();

    while(i < size && blockId > 0 && flags[blockId - 1].load() != 'P')
        continue;

    __syncthreads();

    if (i < size && tid == 0)
        flags[blockId].store('P');
}*/


// Kogge-Stone scan with decoupled-loop back
__global__
void compact_scan(int* image, int size, int *blockNb, cuda::std::atomic<char>* flags, int* predicate){

    __shared__ int blockId;
    if (threadIdx.x == 0)
        blockId = atomicAdd(blockNb, 1);
    __syncthreads();

    extern __shared__ int sdata[];

    constexpr int garbage_val = -27;

    int tid = threadIdx.x;
    int i = blockId * blockDim.x + threadIdx.x;

    if (i < size){
        if (image[i] != garbage_val)
            sdata[tid] = 1;
        else
            sdata[tid] = 0;
    }
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        int data;
        if (i < size && tid + s < blockDim.x){
            data = sdata[tid];
        }
        __syncthreads();
        if (i < size && tid + s < blockDim.x){
            sdata[tid + s] += data;
        }
        __syncthreads();
    }
    __syncthreads();

    if (i < size)
        flags[blockId].store('A');

    while(i < size && blockId > 0 && flags[blockId - 1].load() != 'P')
        continue;

    __syncthreads();

    if (i < size)
        predicate[tid + blockId * blockDim.x] = sdata[tid];

    __syncthreads();

    if (i < size && blockId != 0)
        predicate[tid + blockId * blockDim.x] += predicate[blockId * blockDim.x - 1];

    if (i < size)
        flags[blockId].store('P');
}

__global__
void compact_scatter(int* image, int* predicate, int size, int* output){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int val;
    if (i < size)
        val = image[i];
    __syncthreads();
    if (i < size && val != -27)
        output[predicate[i] - 1] = val;
}

__global__
void map_fixer(int* image, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    if (i % 4 == 0)
        image[i] = image[i] + 1 <= 255 ? image[i] + 1 : 255;
    else if (i % 4 == 1)
        image[i] = image[i] - 5 >= 0 ? image[i] - 5 : 0;
    else if (i % 4 == 2)
        image[i] = image[i] + 3 <= 255 ? image[i] + 3 : 255;
    else if (i % 4 == 3)
        image[i] = image[i] - 8 >= 0 ? image[i] - 8 : 0;
}

__global__
void create_histogram(int* image, int* histogram, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        atomicAdd(&histogram[image[i]], 1);
}

__global__
void scan_hist(int* hist){
    extern __shared__ int sdata[];

    int tid = threadIdx.x;

    sdata[tid] = hist[tid];

    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        int data;
        if (tid + s < blockDim.x){
            data = sdata[tid];
        }
        __syncthreads();
        if (tid + s < blockDim.x){
            sdata[tid + s] += data;
        }
        __syncthreads();
    }
    __syncthreads();

    hist[tid] = sdata[tid];
}

__global__
void apply_equalization(int* image, int* histogram, int size, int cdf_min){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    image[i] = roundf(((histogram[image[i]] - cdf_min) / static_cast<float>(size - cdf_min)) * 255.0f);
}

void fix_image_gpu(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;
    constexpr int blocksize = 256;
    const int gridsize = (to_fix.size() + blocksize - 1) / blocksize;

    int* blockNb;
    hipMalloc(&blockNb, sizeof(int));
    hipMemset(blockNb, 0, sizeof(int));

    int *predicate;
    hipMalloc(&predicate, sizeof(int) * to_fix.size());
    hipMemset(predicate, 0, sizeof(int) * to_fix.size());

    int *histogram;
    hipMalloc(&histogram, sizeof(int) * 256);
    hipMemset(histogram, 0, sizeof(int) * 256);

    cuda::std::atomic<char>* flags;
    hipMalloc(&flags, sizeof(cuda::std::atomic<char>) * gridsize);
    hipMemset(flags, 'X', sizeof(cuda::std::atomic<char>) * gridsize);

    int* image_gpu;
    hipMalloc(&image_gpu, sizeof(int) * to_fix.size());
    hipMemcpy(image_gpu, to_fix.buffer, sizeof(int) * to_fix.size(), hipMemcpyHostToDevice);

    int* clean_image;
    hipMalloc(&clean_image, sizeof(int) * image_size);
    hipMemset(clean_image, 0, sizeof(int) * image_size);

    compact_scan<<<gridsize, blocksize, sizeof(int) * blocksize * 2 + sizeof(int)>>>(image_gpu, to_fix.size(), blockNb, flags, predicate);
    compact_scatter<<<gridsize, blocksize>>>(image_gpu, predicate, to_fix.size(), clean_image);
    map_fixer<<<gridsize, blocksize>>>(clean_image, image_size);
    create_histogram<<<gridsize, blocksize>>>(clean_image, histogram, image_size);
    scan_hist<<<1, blocksize, sizeof(int) * 256 + sizeof(int)>>>(histogram);

    int* final_hist = (int*)calloc(256, sizeof(int));
    hipMemcpy(final_hist, histogram, sizeof(int) * 256, hipMemcpyDeviceToHost);

    auto first_none_zero = std::find_if(final_hist, final_hist + 256, [](auto v) { return v != 0; });
    const int cdf_min = *first_none_zero;

    hipMemcpy(histogram, final_hist, sizeof(int) * 256, hipMemcpyHostToDevice);
    apply_equalization<<<gridsize, blocksize>>>(clean_image, histogram, image_size, cdf_min);
    
    hipMemcpy(to_fix.buffer, clean_image, image_size * sizeof(int), hipMemcpyDeviceToHost);
}