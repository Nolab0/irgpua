#include "hip/hip_runtime.h"
#include "fix_gpu.cuh"
#include "image.hh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <cuda/atomic>

__global__
void compact_scan(int* image, int size, int *blockNb, cuda::std::atomic<char>* flags, int* predicate){

    __shared__ int blockId;
    if (threadIdx.x == 0)
        blockId = atomicAdd(blockNb, 1);
    __syncthreads();

    extern __shared__ int sdata[];

    constexpr int garbage_val = -27;

    int tid = threadIdx.x;
    int i = blockId * blockDim.x + threadIdx.x;

    if (i >= size)
        return;

    if (image[i] != garbage_val)
        sdata[tid] = 1;
    else
        sdata[tid] = 0;
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        int data;
        if (tid + s < blockDim.x){
            data = sdata[tid];
        }
        __syncthreads();
        if (tid + s < blockDim.x){
            sdata[tid + s] += data;
        }
        __syncthreads();
    }
    __syncthreads();

    flags[blockId].store('A');

    while(blockId > 0 && flags[blockId - 1].load() != 'P')
        continue;

    __syncthreads();

    predicate[tid + blockId * blockDim.x] = sdata[tid];

    __syncthreads();

    if (blockId != 0)
        predicate[tid + blockId * blockDim.x] += predicate[blockId * blockDim.x - 1];

    flags[blockId].store('P');
}

__global__
void compact_scatter(int* image, int* predicate, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    if (image[i] != -27)
        image[predicate[i] - 1] = image[i];
}

__global__
void map_fixer(int* image, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    if (i % 4 == 0)
        image[i] = image[i] + 1 <= 255 ? image[i] + 1 : 255;
    else if (i % 4 == 1)
        image[i] = image[i] - 5 >= 0 ? image[i] - 5 : 0;
    else if (i % 4 == 2)
        image[i] = image[i] + 3 <= 255 ? image[i] + 3 : 255;
    else if (i % 4 == 3)
        image[i] = image[i] - 8 >= 0 ? image[i] - 8 : 0;
}

__global__
void create_histogram(int* image, int* histogram, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        atomicAdd(&histogram[image[i]], 1);
}

__global__
void scan_hist(int* hist){
    extern __shared__ int sdata[];

    int tid = threadIdx.x;

    sdata[tid] = hist[tid];

    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        int data;
        if (tid + s < blockDim.x){
            data = sdata[tid];
        }
        __syncthreads();
        if (tid + s < blockDim.x){
            sdata[tid + s] += data;
        }
        __syncthreads();
    }
    __syncthreads();

    hist[tid] = sdata[tid];
}

__global__
void apply_equalization(int* image, int* histogram, int size, int cdf_min){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    image[i] = roundf(((histogram[image[i]] - cdf_min) / static_cast<float>(size - cdf_min)) * 255.0f);
}

void fix_image_gpu(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;
    constexpr int blocksize = 256;
    const int gridsize = (to_fix.size() + blocksize - 1) / blocksize;

    int* blockNb;
    hipMalloc(&blockNb, sizeof(int));
    hipMemset(blockNb, 0, sizeof(int));

    int *predicate;
    hipMalloc(&predicate, sizeof(int) * to_fix.size());
    hipMemset(predicate, 0, sizeof(int) * to_fix.size());

    int *histogram;
    hipMalloc(&histogram, sizeof(int) * 256);
    hipMemset(histogram, 0, sizeof(int) * 256);

    cuda::std::atomic<char>* flags;
    hipMalloc(&flags, sizeof(cuda::std::atomic<char>) * gridsize);
    hipMemset(flags, 'X', sizeof(cuda::std::atomic<char>) * gridsize);

    int* image_gpu;
    hipMalloc(&image_gpu, sizeof(int) * to_fix.size());
    hipMemcpy(image_gpu, to_fix.buffer, sizeof(int) * to_fix.size(), hipMemcpyHostToDevice);

    compact_scan<<<gridsize, blocksize, sizeof(int) * blocksize + sizeof(int)>>>(image_gpu, to_fix.size(), blockNb, flags, predicate);
    compact_scatter<<<gridsize, blocksize>>>(image_gpu, predicate, to_fix.size());
    map_fixer<<<gridsize, blocksize>>>(image_gpu, image_size);
    create_histogram<<<gridsize, blocksize>>>(image_gpu, histogram, image_size);
    scan_hist<<<1, blocksize, sizeof(int) * 256 + sizeof(int)>>>(histogram);

    int* final_hist = (int*)calloc(256, sizeof(int));
    hipMemcpy(final_hist, histogram, sizeof(int) * 256, hipMemcpyDeviceToHost);

    auto first_none_zero = std::find_if(final_hist, final_hist + 256, [](auto v) { return v != 0; });
    const int cdf_min = *first_none_zero;

    hipMemcpy(histogram, final_hist, sizeof(int) * 256, hipMemcpyHostToDevice);
    apply_equalization<<<gridsize, blocksize>>>(image_gpu, histogram, image_size, cdf_min);
    
    hipMemcpy(to_fix.buffer, image_gpu, image_size * sizeof(int), hipMemcpyDeviceToHost);
}