#include "hip/hip_runtime.h"
#include "fix_gpu.cuh"
#include "image.hh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <cuda/atomic>
#include <cub/device/device_reduce.cuh>
#include <cub/util_allocator.cuh>

__global__
void kernel_reduce(int* buffer, int* total, int size) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    if (i < size) {
        int val = buffer[i];
        if (i + blockDim.x < size) {
            val += buffer[i + blockDim.x];
        }
        sdata[tid] = val;
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) atomicAdd(&total[0], sdata[0]);
}

int reduce_gpu(Image& image)
{
    const int image_size = image.width * image.height;
    constexpr int blocksize = 256;
    const int gridsize = (image_size + blocksize - 1) / blocksize;

    int* image_gpu;
    hipMalloc(&image_gpu, sizeof(int) * image_size);
    hipMemcpy(image_gpu, image.buffer, sizeof(int) * image_size, hipMemcpyHostToDevice);

    int* total;
    hipMalloc(&total, sizeof(int));
    hipMemset(total, 0, sizeof(int));

	kernel_reduce<<<gridsize, blocksize, sizeof(int) * blocksize>>>(image_gpu, total, image_size);

    int *total_local = (int*)malloc(sizeof(int));
    hipMemcpy(total_local, total, sizeof(int), hipMemcpyDeviceToHost);

    return *total_local;
}

int reduce_gpu_industrial(Image& image) {
    const int image_size = image.width * image.height;

    int* image_gpu;
    hipMalloc(&image_gpu, sizeof(int) * image_size);
    hipMemcpy(image_gpu, image.buffer, sizeof(int) * image_size, hipMemcpyHostToDevice);

    int* total;
    hipMalloc(&total, sizeof(int));
    hipMemset(total, 0, sizeof(int));
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, image_gpu, total, image_size);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, image_gpu, total, image_size);

    hipFree(d_temp_storage);
    hipFree(image_gpu);

    int *total_local = (int*)malloc(sizeof(int));
    hipMemcpy(total_local, total, sizeof(int), hipMemcpyDeviceToHost);

    return *total_local;
}