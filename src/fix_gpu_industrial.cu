#include "hip/hip_runtime.h"
#include "fix_gpu_industrial.cuh"
#include "image.hh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <cuda/atomic>
#include <hipcub/hipcub.hpp>

__global__
void build_predicate(int* image, int size, int* predicate){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
     __syncthreads();
    if (image[i] != -27)
        predicate[i] = 1;
}

__global__
void compact_scatter(int* image, int* predicate, int size, int* output){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int val;
    if (i < size)
        val = image[i];
    __syncthreads();
    if (i < size && val != -27)
        output[predicate[i]] = val;
}

__global__
void map_fixer(int* image, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    if (i % 4 == 0)
        image[i] = image[i] + 1 <= 255 ? image[i] + 1 : 255;
    else if (i % 4 == 1)
        image[i] = image[i] - 5 >= 0 ? image[i] - 5 : 0;
    else if (i % 4 == 2)
        image[i] = image[i] + 3 <= 255 ? image[i] + 3 : 255;
    else if (i % 4 == 3)
        image[i] = image[i] - 8 >= 0 ? image[i] - 8 : 0;
}

__global__
void create_histogram(int* image, int* histogram, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        atomicAdd(&histogram[image[i]], 1);
}

__global__
void scan_hist(int* hist){
    extern __shared__ int sdata[];

    int tid = threadIdx.x;

    sdata[tid] = hist[tid];

    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        int data;
        if (tid + s < blockDim.x){
            data = sdata[tid];
        }
        __syncthreads();
        if (tid + s < blockDim.x){
            sdata[tid + s] += data;
        }
        __syncthreads();
    }
    __syncthreads();

    hist[tid] = sdata[tid];
}

__global__
void apply_equalization(int* image, int* histogram, int size, int cdf_min){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    image[i] = roundf(((histogram[image[i]] - cdf_min) / static_cast<float>(size - cdf_min)) * 255.0f);
}

void fix_image_gpu_industrial(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;
    constexpr int blocksize = 256;
    const int gridsize = (to_fix.size() + blocksize - 1) / blocksize;

    int *predicate;
    hipMalloc(&predicate, sizeof(int) * to_fix.size());
    hipMemset(predicate, 0, sizeof(int) * to_fix.size());

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, predicate, predicate, image_size);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    int *histogram;
    hipMalloc(&histogram, sizeof(int) * 256);
    hipMemset(histogram, 0, sizeof(int) * 256);

    int* image_gpu;
    hipMalloc(&image_gpu, sizeof(int) * to_fix.size());
    hipMemcpy(image_gpu, to_fix.buffer, sizeof(int) * to_fix.size(), hipMemcpyHostToDevice);

    int* clean_image;
    hipMalloc(&clean_image, sizeof(int) * image_size);
    hipMemset(clean_image, 0, sizeof(int) * image_size);

    build_predicate<<<gridsize, blocksize>>>(image_gpu, to_fix.size(), predicate);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, predicate, predicate, to_fix.size());
    compact_scatter<<<gridsize, blocksize>>>(image_gpu, predicate, to_fix.size(), clean_image);

    map_fixer<<<gridsize, blocksize>>>(clean_image, image_size);
    create_histogram<<<gridsize, blocksize>>>(clean_image, histogram, image_size);
    scan_hist<<<1, blocksize, sizeof(int) * 256 + sizeof(int)>>>(histogram);

    int* final_hist = (int*)calloc(256, sizeof(int));
    hipMemcpy(final_hist, histogram, sizeof(int) * 256, hipMemcpyDeviceToHost);

    auto first_none_zero = std::find_if(final_hist, final_hist + 256, [](auto v) { return v != 0; });
    const int cdf_min = *first_none_zero;

    hipMemcpy(histogram, final_hist, sizeof(int) * 256, hipMemcpyHostToDevice);
    apply_equalization<<<gridsize, blocksize>>>(clean_image, histogram, image_size, cdf_min);
    
    hipMemcpy(to_fix.buffer, clean_image, image_size * sizeof(int), hipMemcpyDeviceToHost);
}