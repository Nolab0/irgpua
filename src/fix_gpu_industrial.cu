#include "hip/hip_runtime.h"
#include "fix_gpu_industrial.cuh"
#include "image.hh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <cuda/atomic>
#include <hipcub/hipcub.hpp>

__global__
void build_predicate(int* image, int size, int* predicate){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
     __syncthreads();
    if (image[i] != -27)
        predicate[i] = 1;
}

__global__
void compact_scatter_industrial(int* image, int* predicate, int size, int* output){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int val;
    if (i < size)
        val = image[i];
    __syncthreads();
    if (i < size && val != -27)
        output[predicate[i]] = val;
}

__global__
void map_fixer_industrial(int* image, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    if (i % 4 == 0)
        image[i] = image[i] + 1 <= 255 ? image[i] + 1 : 255;
    else if (i % 4 == 1)
        image[i] = image[i] - 5 >= 0 ? image[i] - 5 : 0;
    else if (i % 4 == 2)
        image[i] = image[i] + 3 <= 255 ? image[i] + 3 : 255;
    else if (i % 4 == 3)
        image[i] = image[i] - 8 >= 0 ? image[i] - 8 : 0;
}

__global__
void create_histogram_industrial(int* image, int* histogram, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        atomicAdd(&histogram[image[i]], 1);
}

__global__
void scan_hist_industrial(int* hist){
    extern __shared__ int sdata[];

    int tid = threadIdx.x;

    sdata[tid] = hist[tid];

    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        int data;
        if (tid + s < blockDim.x){
            data = sdata[tid];
        }
        __syncthreads();
        if (tid + s < blockDim.x){
            sdata[tid + s] += data;
        }
        __syncthreads();
    }
    __syncthreads();

    hist[tid] = sdata[tid];
}

__global__
void apply_equalization_industrial(int* image, int* histogram, int size, int cdf_min){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    image[i] = roundf(((histogram[image[i]] - cdf_min) / static_cast<float>(size - cdf_min)) * 255.0f);
}

void fix_image_gpu_industrial(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;
    constexpr int blocksize = 256;
    const int gridsize = (to_fix.size() + blocksize - 1) / blocksize;

    int *predicate;
    hipMalloc(&predicate, sizeof(int) * to_fix.size());
    hipMemset(predicate, 0, sizeof(int) * to_fix.size());

    void *d_temp_storage_compact = nullptr;
    size_t temp_storage_bytes_compact = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage_compact, temp_storage_bytes_compact, predicate, predicate, image_size);

    hipMalloc(&d_temp_storage_compact, temp_storage_bytes_compact);

    int *histogram;
    hipMalloc(&histogram, sizeof(int) * 256);
    hipMemset(histogram, 0, sizeof(int) * 256);

    void *d_temp_storage_hist = nullptr;
    size_t temp_storage_bytes_hist = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage_hist, temp_storage_bytes_hist, histogram, histogram, image_size);

    hipMalloc(&d_temp_storage_hist, temp_storage_bytes_hist);

    int* image_gpu;
    hipMalloc(&image_gpu, sizeof(int) * to_fix.size());
    hipMemcpy(image_gpu, to_fix.buffer, sizeof(int) * to_fix.size(), hipMemcpyHostToDevice);

    int* clean_image;
    hipMalloc(&clean_image, sizeof(int) * image_size);
    hipMemset(clean_image, 0, sizeof(int) * image_size);

    build_predicate<<<gridsize, blocksize>>>(image_gpu, to_fix.size(), predicate);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage_compact, temp_storage_bytes_compact, predicate, predicate, to_fix.size());
    compact_scatter_industrial<<<gridsize, blocksize>>>(image_gpu, predicate, to_fix.size(), clean_image);

    map_fixer_industrial<<<gridsize, blocksize>>>(clean_image, image_size);
    create_histogram_industrial<<<gridsize, blocksize>>>(clean_image, histogram, image_size);
    hipcub::DeviceScan::InclusiveSum(d_temp_storage_hist, temp_storage_bytes_hist, histogram, histogram, 256);

    int* final_hist = (int*)calloc(256, sizeof(int));
    hipMemcpy(final_hist, histogram, sizeof(int) * 256, hipMemcpyDeviceToHost);

    auto first_none_zero = std::find_if(final_hist, final_hist + 256, [](auto v) { return v != 0; });
    const int cdf_min = *first_none_zero;

    hipMemcpy(histogram, final_hist, sizeof(int) * 256, hipMemcpyHostToDevice);
    apply_equalization_industrial<<<gridsize, blocksize>>>(clean_image, histogram, image_size, cdf_min);
    
    hipMemcpy(to_fix.buffer, clean_image, image_size * sizeof(int), hipMemcpyDeviceToHost);
}